#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <cmath>

using namespace std;
const unsigned SIZE = 40;
const unsigned SCALE = 100;

void matrixSet(int* array, unsigned size) {
    srand(time(nullptr));

    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            *(array + i * size + j) = rand() % SCALE;
        }
    }
}


void matrixPrint(const int* matrix, unsigned size) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j)
            cout << *(matrix + j + i * size) << ' ';
        cout << endl;
    }
    cout << endl;
}

__global__ void gpuDeterminant(const int* matrix, unsigned size, long long* diagonals) {
    unsigned threadNumber = blockIdx.x * blockDim.x + threadIdx.x;  // Номер текущего потока

    /* Для матрицы размера N имеем N*2 дианогалей:
     * N положительных, и N отрицательных. */
    if (threadNumber < (size * 2)) {

        long long temp = 1;
        unsigned ind = threadNumber;

        if (threadNumber < size) {
            /* Положительная диагональ. */
            for (unsigned i = 0; i < size; i++) {
                temp *= (*(matrix + i * size + ind));
                ind = (ind + 1) % size;
            }
        }   else    {
            ind = threadNumber % size;
            /* Отрицательная диагональ. */
            for (unsigned i = 0; i < size; i++) {
                temp *= (*(matrix + i * size + ind));
                ind = ((ind + size - 1) % size);
            }
            temp *= -1;
        }
        /* Фиксируем полученное значение. */
        *((long long*)((char*)diagonals + threadNumber * sizeof(long long))) = temp;
    }
}

__global__ void inverseByMinors(const int* matrix, unsigned size, double* inverseMatrix,
                                long long initialDeterminant) {
    /* Номер текущего потока. */
    unsigned threadNumber = blockIdx.x * blockDim.x + threadIdx.x;
    /* Текущее количество элементов в миноре. */
    unsigned count = 0;
    /* Номера вычеркнутых столбца и ряда в миноре. */
    const unsigned row = threadNumber / size;
    const unsigned col = threadNumber % size;
    /* Размер минора. */
    const unsigned minorSize = size - 1;

    int* minorMatrix;
    hipMalloc((void**)&minorMatrix, sizeof(int) * minorSize * minorSize);

    if(minorMatrix == nullptr){
        printf("Not enough memory for minor matrix.\n");
        return;
    }

    /* Собираем минор из элементов основной матрицы. */
    for (int i = 0; i < size; i++)
        for (int j = 0; j < size; j++)
            if (i != row && j != col) {
                minorMatrix[count] = *(matrix + i * size + j);
                count++;
            }

    double minorDeterminant = 0;
    unsigned positiveInd = 0;
    unsigned negativeInd = minorSize - 1;

    /* Вычисляем определитель минора. */
    for (int i = 0; (minorSize != 2 && i < minorSize) || (minorSize == 2 && i < 1); i++) {
        long long temp = 1;
        for (int j = 0; j < minorSize; j++) {
            temp *= *(minorMatrix + j * minorSize + positiveInd);
            positiveInd = (positiveInd + 1) % minorSize;
        }
        positiveInd = (positiveInd + 1) % minorSize;
        minorDeterminant += temp;

        temp = 1;
        for (int j = 0; j < minorSize; j++) {
            temp *= *(minorMatrix + j * minorSize + negativeInd);
            negativeInd = ((negativeInd + minorSize - 1) % minorSize);
        }
        negativeInd = ((negativeInd + minorSize - 1) % minorSize);
        minorDeterminant -= temp;
    }
    hipFree(minorMatrix);

    unsigned degree = (row + col) % 2;
    if (degree)
        minorDeterminant *= -1;

    /* Помещаем полученный элемент в обратную матрицу. */
    *(inverseMatrix + col * size + row) = minorDeterminant / initialDeterminant;
}

__global__ void gpuPrint(const double* matrix, const unsigned size){
    for(int i = 0; i < size; i++){
        for(int j = 0; j < size; j++)
            printf("%f ", *(matrix + i * size + j));
        printf("\n");
    }
    printf("\n");
}

__host__ int main() {
    int matrix[SIZE * SIZE];
    matrixSet(matrix, SIZE);
    matrixPrint(matrix, SIZE);

    /* Копируем матрицу на видеокарту. */
    int* matrixGpu;
    hipMalloc((void**)&matrixGpu, sizeof(int) * SIZE * SIZE);
    hipMemcpy(matrixGpu, &matrix, SIZE * SIZE * sizeof(int), hipMemcpyHostToDevice);

    long long temp = 0;
    long long determinant = 0;

    /* Вычисляем определитель исходной матрицы. */
    long long* determinantDiagonals;
    hipMalloc((void**)&determinantDiagonals, (SIZE * 2 * sizeof(long long)));
    gpuDeterminant <<<SIZE * 2, 1 >>> (matrixGpu, SIZE, determinantDiagonals);
    hipDeviceSynchronize();
    for (int i = 0; i < SIZE * 2; i++) {
        hipMemcpy(&temp, determinantDiagonals + i, sizeof(long long), hipMemcpyDeviceToHost);
        determinant += temp;
    }
    cout << "Determinant = " << determinant << endl << endl;

    hipFree(determinantDiagonals);

    double* inverseMatrixGpu;
    hipMalloc((void**)&inverseMatrixGpu, sizeof(double) * SIZE * SIZE);

    if(inverseMatrixGpu == nullptr){
        cout << "Not enough memory for inverse matrix." << endl;
        return 0;
    }

    /* Вычисляем обратную матрицу, если она существует. */
    if (determinant != 0) {
        inverseByMinors<<<SIZE, SIZE>>> (matrixGpu, SIZE, inverseMatrixGpu, determinant);
        hipDeviceSynchronize();
        cout << "Inverse matrix:" << endl << endl;
        gpuPrint<<<1, 1>>>(inverseMatrixGpu, SIZE);
    }   else {
        cout << "Inverse matrix not exists." << endl;
        hipFree(inverseMatrixGpu);
        hipFree(matrixGpu);
    }
    return 0;
}
